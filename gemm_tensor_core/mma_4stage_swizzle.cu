// 2 mma + pipeline + double threading + smem crosswise


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

const int bm = 128;
const int bn = 128;
const int bk = 32;

const int wm = 64;
const int wn = 32;
const int wk = 16;

const int wmma_m = 16;
const int wmma_n = 16;
// const int wmmaK = 16;

__device__ void loadSmemA(half *smem, half *A, int M, int K, int ko) {
    // load 128 * 32
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 128 + ty * 32 + tx;
    for (int i = 0; i < 2; ++i) {
        int logic_row = i * 64 + tid / 4;
        int logic_col = tid % 4 * 8;
        int row = i * 32 + tid / 8;
        int col = tid % 8 * 8;
        col = col ^ (((row & 3) << 3));
        void *ptr = (void *)(smem + row * 64 + col);
        uint32_t smem_ptr;

        asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
            "%0, smem_ptr; }\n"
            : "=r"(smem_ptr)
            : "l"(ptr));

        asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_ptr),
                        "l"(&A[(by * 128 + logic_row) * K + (ko * bk + logic_col)]),
                        "n"(16));
    }
}

__device__ void loadSmemB(half *smem, half *B, int N, int K, int ko) {
  // load 128 * 32
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
  for (int i = 0; i < 2; ++i) {
    int logic_row = i * 64 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 32 + tid / 8;
    int col = tid % 8 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;

    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));

    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_ptr),
                 "l"(&B[(bx * 128 + logic_row) * K + (ko * bk + logic_col)]),
                 "n"(16));
  }
}

__device__ void loadSmemC(float *smem, half *C, int M, int N) {
  // load 128 * 128
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
  for (int i = 0; i < 64; ++i) {
    int row = i * 2 + tid / 128;
    int col = tid % 128;
    smem[row * 128 + col] = (float)(C[(by * 128 + row) * N + bx * 128 + col]);
  }
}

__device__ void storeSmemC(half *C, float *smem, int M, int N) {
  // load 128 * 128
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
  for (int i = 0; i < 64; ++i) {
    int row = i * 2 + tid / 128;
    int col = tid % 128;
    (C[(by * 128 + row) * N + bx * 128 + col]) = (half)smem[row * 128 + col];
  }
}

__device__ void loadFragA(unsigned int *frag, half *smem, int ki) {
  // frag: [j, k]: [2, 2]
  // load 64x16
  int tx = threadIdx.x;
  int tz = threadIdx.z;
  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 2; ++j) {
      for (int k = 0; k < 2; ++k) {
        int row = tz * 64 + i * 16 + j * 8 + tx / 4;
        int col = ki * wk + k * 8 + tx % 4 * 2;
        col = row % 2 * 32 + col;
        row = row / 2;
        col = col ^ ((row & 3) << 3);
        unsigned int *ptr =
            reinterpret_cast<unsigned int *>(smem + row * 64 + col);
        frag[i * 4 + j * 2 + k] = ptr[0];
      }
    }
  }
}

__device__ void loadFragB(unsigned int *frag, half *smem, int ki) {
  // frag: [j, k]: []
  // load 32x16
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 2; ++j) {
      for (int k = 0; k < 2; ++k) {
        int row = ty * 32 + i * 16 + j * 8 + tx / 4;
        int col = ki * wk + k * 8 + tx % 4 * 2;
        col = row % 2 * 32 + col;
        row = row / 2;
        col = col ^ ((row & 3) << 3);
        unsigned int *ptr =
            reinterpret_cast<unsigned int *>(smem + row * 64 + col);
        frag[i * 4 + j * 2 + k] = ptr[0];
      }
    }
  }
}

__device__ void storeAccum(float *ptr, float *frag) {
  // frag [r, c, _]: [2, 2, 2]
  // store 64x32
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int row = tz * 64 + tx / 4;
  int col = ty * 32 + tx % 4 * 2;
  // float *dst = ptr + row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16) + row %
  // 16 * 16 + col % 16;
  float *dst = ptr + row * 128 + col;
  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 2; ++j) {
      dst[0] = frag[i * 16 + j * 8 + 0 * 4 + 0 * 2];
      dst[1] = frag[i * 16 + j * 8 + 0 * 4 + 0 * 2 + 1];

      dst[0 + 8] = frag[i * 16 + j * 8 + 0 * 4 + 1 * 2];
      dst[1 + 8] = frag[i * 16 + j * 8 + 0 * 4 + 1 * 2 + 1];

      dst[0 + 8 * 128] = frag[i * 16 + j * 8 + 1 * 4 + 0 * 2];
      dst[1 + 8 * 128] = frag[i * 16 + j * 8 + 1 * 4 + 0 * 2 + 1];

      dst[0 + 8 * 128 + 8] = frag[i * 16 + j * 8 + 1 * 4 + 1 * 2];
      dst[1 + 8 * 128 + 8] = frag[i * 16 + j * 8 + 1 * 4 + 1 * 2 + 1];

      dst += 16;
    }
    dst += 16 * 128 - 16 * 2;
  }
}

__device__ void mmaSync(unsigned int *fragA, unsigned int *fragB,
                        float *accum) {
  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[0]), "=f"(accum[1]), "=f"(accum[4]), "=f"(accum[5])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[0]), "r"(fragB[1]), "f"(accum[0]), "f"(accum[1]),
                 "f"(accum[4]), "f"(accum[5]));

  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[2]), "=f"(accum[3]), "=f"(accum[6]), "=f"(accum[7])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[2]), "r"(fragB[3]), "f"(accum[2]), "f"(accum[3]),
                 "f"(accum[6]), "f"(accum[7]));
}

__global__ void matmul(half *A, half *B, half *C, int M, int N, int K, float alpha, float beta) {
  // A is row-major
  // B is col-major
  // 128 threads [x, y, z] = [32, 2, 2]
  // threadblock mma: 128x128x32
  // warp mma: 64x64x16
  extern __shared__ uint8_t shared_storage[];
  half *SA1 = reinterpret_cast<half *>(shared_storage);
  half *SA2 = SA1 + bm * bk;
  half *SA3 = SA2 + bm * bk;
  half *SA4 = SA3 + bm * bk;
  half *SB1 = SA4 + bm * bk;
  half *SB2 = SB1 + bn * bk;
  half *SB3 = SB2 + bn * bk;
  half *SB4 = SB3 + bn * bk;
  float *SC = reinterpret_cast<float *>(shared_storage);

  unsigned int FragA[4 * 4];      // [4, 4]
  unsigned int FragB[2 * 4];      // [2, 4]
  float Accum[4 * 2 * 8] = {0.0}; // [4, 2, 8]

  // prologue
  loadSmemA(SA1, A, M, K, 0);
  loadSmemB(SB1, B, N, K, 0);
  asm volatile("cp.async.commit_group;\n" ::);

  loadSmemA(SA2, A, M, K, 1);
  loadSmemB(SB2, B, N, K, 1);
  asm volatile("cp.async.commit_group;\n" ::);

  loadSmemA(SA3, A, M, K, 2);
  loadSmemB(SB3, B, N, K, 2);
  asm volatile("cp.async.commit_group;\n" ::);

  for (int ko = 0; ko < K / bk - 4; ko += 4) {
    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 3 < K / bk) {
      loadSmemA(SA4, A, M, K, ko + 3);
      loadSmemB(SB4, B, N, K, ko + 3);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA1, ki);
      loadFragB(FragB, SB1, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 4 < K / bk) {
      loadSmemA(SA1, A, M, K, ko + 4);
      loadSmemB(SB1, B, N, K, ko + 4);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA2, ki);
      loadFragB(FragB, SB2, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 5 < K / bk) {
      loadSmemA(SA2, A, M, K, ko + 5);
      loadSmemB(SB2, B, N, K, ko + 5);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA3, ki);
      loadFragB(FragB, SB3, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 6 < K / bk) {
      loadSmemA(SA3, A, M, K, ko + 6);
      loadSmemB(SB3, B, N, K, ko + 6);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA4, ki);
      loadFragB(FragB, SB4, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }
  }

  // the last 4 iterations
  {
    int ko = (K / bk / 4 - 1) * 4;
    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 3 < K / bk) {
      loadSmemA(SA4, A, M, K, ko + 3);
      loadSmemB(SB4, B, N, K, ko + 3);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA1, ki);
      loadFragB(FragB, SB1, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 4 < K / bk) {
      loadSmemA(SA1, A, M, K, ko + 4);
      loadSmemB(SB1, B, N, K, ko + 4);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA2, ki);
      loadFragB(FragB, SB2, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(1));
    __syncthreads();
    if (ko + 5 < K / bk) {
      loadSmemA(SA2, A, M, K, ko + 5);
      loadSmemB(SB2, B, N, K, ko + 5);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA3, ki);
      loadFragB(FragB, SB3, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(0));
    __syncthreads();
    if (ko + 6 < K / bk) {
      loadSmemA(SA3, A, M, K, ko + 6);
      loadSmemB(SB3, B, N, K, ko + 6);
    }
    for (int ki = 0; ki < bk / wk; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA4, ki);
      loadFragB(FragB, SB4, ki);
      for (int mii = 0; mii < wm / wmma_m; mii += 1) {
        for (int nii = 0; nii < wn / wmma_n; nii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }
  }
  storeAccum(SC, Accum);
  __syncthreads();
  storeSmemC(C, SC, M, N);
}