// wmma + pipeline

#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>

using namespace nvcuda;

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void loadSmemA(half *smem, half *A, int M, int K, int k) {
    // load 128 * 32
    const int by = blockIdx.y;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int row = (i << 5) + (tid >> 2); // 1 thread load 128-bit, 4 threads per row
        const int col = (lane_id >> 2) << 3; // 128-bit per thread, aka 8 half per thread

        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        void *ptr = reinterpret_cast<void *>(smem + (row_o << 9) + (col_o << 8) + (row_i << 4) + col_i);
        uint32_t smem_ptr;

        asm(
            "{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, smem_ptr; }\n"
            : "=r"(smem_ptr)
            : "l"(ptr)
        );

        asm volatile(
            "cp.async.cg.shared.global [%0], [%1], %2;\n"
            :
            : "r"(smem_ptr), "l"(&A[(by * bm + row) * K + (k * bk + col)]), "n"(16)
        );
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void loadSmemB(half *smem, half *B, int N, int K, int k) {
    // load 128 * 32
    const int bx = blockIdx.x;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    // const int tid = warp_y * 64 + warp_x * 32 + lane_id;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int row = (i << 5) + (tid >> 2); // 1 thread load 128-bit, 4 threads per row
        const int col = (lane_id >> 2) << 3; // 128-bit per thread, aka 8 half per thread

        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        void *ptr = reinterpret_cast<void *>(smem + (row_o << 9) + (col_o << 8) + (row_i << 4) + col_i);
        uint32_t smem_ptr;

        asm(
            "{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, smem_ptr; }\n"
            : "=r"(smem_ptr)
            : "l"(ptr)
        );

        asm volatile(
            "cp.async.cg.shared.global [%0], [%1], %2;\n" 
            :
            : "r"(smem_ptr), "l"(&B[(bx * bn + row) * K + (k * bk + col)]), "n"(16)
        );
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void loadSmemC(float *smem, half *C, int M, int N) {
    // load 128 * 128
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < bm; ++i) {
        const int row = i;
        const int col = tid;

        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        smem[(row_o << 9) + (col_o << 8) + (row_i << 4) + col_i] = 
            static_cast<float>(C[(by * bm + row) * N + bx * bn + col]);
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void storeSmemC(half *C, float *smem, int M, int N) {
    // load 128 * 128
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int lane_id = threadIdx.x;
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;
    const int tid = (warp_y << 6) + (warp_x << 5) + lane_id;

    #pragma unroll
    for (int i = 0; i < bm; ++i) {
        const int row = i;
        const int col = tid;

        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        const int row_o = row >> 4;
        const int col_o = col >> 4;
        const int row_i = row & 15;
        const int col_i = col & 15;
        C[(by * bm + row) * N + bx * bm + col] = 
            static_cast<half>(smem[(row_o << 9) + (col_o << 8) + (row_i << 4) + col_i]);
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void loadFragA(
    wmma::fragment<wmma::matrix_a, wmma_m, wmma_n, wmma_k, half, wmma::row_major> *frag, 
    half *smem, 
    int k
) {
    // load 64x16
    const int warp_y = threadIdx.z;
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int row = (warp_y << 6) + (i << 4);
        const int col = k * wk;
        nvcuda::wmma::load_matrix_sync(
            frag[i], 
            smem + ((row >> 4) << 9) + ((col >> 4) << 8), 
            16
        );
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void loadFragB(
    wmma::fragment<wmma::matrix_b, wmma_m, wmma_n, wmma_k, half, wmma::col_major> *frag, 
    half *smem, 
    int ki
) {
    // load 64x16
    int warp_x = threadIdx.y;
    for (int i = 0; i < 4; ++i) {
        const int row = (warp_x << 6) + (i << 4);
        const int col = ki * wk;
        nvcuda::wmma::load_matrix_sync(
            frag[i], 
            smem + ((row >> 4) << 9) + ((col >> 4) << 8), 
            16
        );
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void storeAccum(
    float *ptr, 
    wmma::fragment<wmma::accumulator, wmma_m, wmma_n, wmma_k, float> *frag
) {
    // store 64x64
    const int warp_x = threadIdx.y;
    const int warp_y = threadIdx.z;

    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            const int row = (warp_y << 6) + (i << 4);
            const int col = (warp_x << 6) + (j << 4);

            // laoyut: [8, 8, 16, 16]
            nvcuda::wmma::store_matrix_sync(
                ptr + ((row >> 4) << 9) + ((col >> 4) << 8), 
                frag[(i << 2) + j], 16, 
                nvcuda::wmma::mem_row_major
            );
        }
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void warpMma(
    wmma::fragment<wmma::matrix_a, wmma_m, wmma_n, wmma_k, half, wmma::row_major> *frag_a, 
    wmma::fragment<wmma::matrix_b, wmma_m, wmma_n, wmma_k, half, wmma::col_major> *frag_b, 
    wmma::fragment<wmma::accumulator, wmma_m, wmma_n, wmma_k, float> *accum,
    half *SA, half *SB,
    const int warp_iters,
    const int frags_m, const int frags_n
) {
    #pragma unroll
    for (int k = 0; k < warp_iters; ++k) {
        // 64x64x16 mma for each warp
        loadFragA(frag_a, SA, k);
        loadFragB(frag_b, SB, k);

        #pragma unroll
        for (int i = 0; i < frags_m; ++i) {
            #pragma unroll
            for (int j = 0; j < frags_n; ++j) {
                // 16x16x16 for each wmma
                nvcuda::wmma::mma_sync(
                    accum[i * frags_n + j], 
                    frag_a[i], frag_b[j], 
                    accum[i * frags_n + j]
                );
            }
        }
    }
}

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__device__ __forceinline__ void loadSmemAndCommit(
    half *SA, half *SB, 
    half *A, half *B, 
    const int k, 
    const int M, const int N, const int K
) {
    loadSmemA(SA, A, M, K, k);
    loadSmemB(SB, B, N, K, k);
    asm volatile("cp.async.commit_group;\n" ::);
}

/*
A is row-major
B is col-major
128 threads [x, y, z] = [32, 2, 2]
threadblock mma: 128x128x32
warp mma: 64x64x16
*/

template <
    int bm = 128, int bn = 128, int bk = 32, 
    int wm = 64, int wn = 64, int wk = 16,
    int wmma_m = 16, int wmma_n = 16, int wmma_k = 16
>
__global__ void matmul(
    half *A, half *B, half *C, 
    const int M, const int N, const int K, 
    const float alpha, const float beta
) {
    extern __shared__ char shared_storage[];
    half *SA1 = reinterpret_cast<half *>(shared_storage);
    half *SA2 = SA1 + bm * bk;
    half *SB1 = SA2 + bm * bk;
    half *SB2 = SB1 + bn * bk;
    float *SC = reinterpret_cast<float *>(shared_storage);

    const int frags_m = wm / wmma_m;
    const int frags_n = wn / wmma_n;
    wmma::fragment<wmma::matrix_a, wmma_m, wmma_n, wmma_k, half, wmma::row_major> frag_a[frags_m];
    wmma::fragment<wmma::matrix_b, wmma_m, wmma_n, wmma_k, half, wmma::col_major> frag_b[frags_n];
    wmma::fragment<wmma::accumulator, wmma_m, wmma_n, wmma_k, float> accum[frags_m * frags_n];

    for (int i = 0; i < frags_m * frags_n; ++i) {
        wmma::fill_fragment(accum[i], 0.0);
    }
    
    // prologue
    loadSmemAndCommit(SA1, SB1, A, B, 0, M, N, K);

    const int block_iters = K / bk;
    const int warp_iters = bk / wk;

    #pragma unroll
    for (int ko = 0; ko + 2 < block_iters; ko += 2) {
        loadSmemAndCommit(SA2, SB2, A, B, ko + 1, M, N, K);
        asm volatile("cp.async.wait_group %0;\n" ::"n"(1));
        __syncthreads();
        warpMma(frag_a, frag_b, accum, SA1, SB1, warp_iters, frags_m, frags_n);

        loadSmemAndCommit(SA1, SB1, A, B, ko + 2, M, N, K);
        asm volatile("cp.async.wait_group %0;\n" ::"n"(1));
        __syncthreads();
        warpMma(frag_a, frag_b, accum, SA2, SB2, warp_iters, frags_m, frags_n);
    }

    {
        int ko = (block_iters / 2 - 1) * 2;

        if (ko < block_iters) {
            warpMma(frag_a, frag_b, accum, SA1, SB1, warp_iters, frags_m, frags_n);
        }
        if (ko + 1 < block_iters) {
            warpMma(frag_a, frag_b, accum, SA2, SB2, warp_iters, frags_m, frags_n);
        }
    }

    storeAccum(SC, accum);
    __syncthreads();
    storeSmemC(C, SC, M, N);
}

template __global__ void matmul<128, 128, 32, 64, 64, 16, 16, 16, 16>(
    half *A, half *B, half *C, 
    const int M, const int N, const int K, 
    const float alpha, const float beta
);