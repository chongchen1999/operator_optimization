#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

const int N = 1 << 25; // 2^25 elements
const int iterations = 5000;

__device__ void warp_reduce(volatile int *shared_data, const int &tid) {
    shared_data[tid] += shared_data[tid + 32];
    shared_data[tid] += shared_data[tid + 16];
    shared_data[tid] += shared_data[tid + 8];
    shared_data[tid] += shared_data[tid + 4];
    shared_data[tid] += shared_data[tid + 2];
    shared_data[tid] += shared_data[tid + 1];
}

template <int block_size>
__global__ void sum_kernel(int *data, int *partial_sums) {
    __shared__ int shared_data[block_size];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int offset = blockDim.x * gridDim.x;

    int sum = 0;
    int i = tid;

    // Unroll the first for-loop
    while (i < N) {
        sum += data[i];
        i += offset;
    }

    shared_data[threadIdx.x] = sum;
    __syncthreads();

    // Unroll the second for-loop
    if (blockDim.x >= 512) {
        if (threadIdx.x < 256) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + 256];
        }
        __syncthreads();
    }

    if (blockDim.x >= 256) {
        if (threadIdx.x < 128) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + 128];
        }
        __syncthreads();
    }

    if (blockDim.x >= 128) {
        if (threadIdx.x < 64) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + 64];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        warp_reduce(shared_data, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        partial_sums[blockIdx.x] = shared_data[0];
    }
}

void get_sum(const int *data, const int &N, int &sum) {
    for (int i = 0; i < N; ++i) {
        sum += data[i];
    }
}

int main() {
    std::srand(static_cast<unsigned>(std::time(nullptr)));
    int *host_data = (int *)malloc(N * sizeof(int));
    int cpu_sum = 0;
    for (int i = 0; i < N; ++i) {
        int random_int = std::rand() % 57;
        host_data[i] = random_int;
        cpu_sum += random_int;
    }
    printf("CPU sum: %d\n", cpu_sum);

    int *device_data;
    hipMalloc(&device_data, N * sizeof(int));

    constexpr int grid_size = 2048;
    constexpr int block_size = 256;

    dim3 block(block_size);
    dim3 grid(grid_size);

    int *host_partial_sums = (int *)malloc(grid_size * sizeof(int));
    int *device_partial_sums;
    hipMalloc(&device_partial_sums, grid_size * sizeof(int));

    hipMemcpy(device_data, host_data, N * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iterations; ++i) {
        sum_kernel<block_size><<<grid, block>>>(device_data, device_partial_sums);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds / iterations);

    hipMemcpy(host_partial_sums, device_partial_sums, grid_size * sizeof(int), hipMemcpyDeviceToHost);
    int gpu_sum = 0;
    get_sum(host_partial_sums, grid_size, gpu_sum);
    if (cpu_sum != gpu_sum) {
        printf("Error: %d != %d\n", cpu_sum, gpu_sum);
    } else {
        printf("Success!\n");
    }

    // Calculate Bandwidth
    float total_data_transferred = N * sizeof(int) + grid_size * sizeof(int); // in bytes
    float average_time_per_iteration = milliseconds / iterations / 1000; // in seconds
    float bandwidth = total_data_transferred / average_time_per_iteration / (1 << 30); // in GB/s

    printf("Bandwidth: %f GB/s\n", bandwidth);

    hipFree(device_data);
    hipFree(device_partial_sums);
    free(host_data);
    free(host_partial_sums);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}